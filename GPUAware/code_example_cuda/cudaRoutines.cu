#include "hip/hip_runtime.h"
#include "cudaRoutines.h"

__global__ void __cuda_increment(int offset, float *vec, float val)
{
    vec[blockIdx.x + offset] += val;
}

void cuda_increment(int M, int N, int offset, float *vec, float val)
{
    __cuda_increment<<<M, N>>>(offset, vec, val);
}
